#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>

#include "draw/host_window.h"

//#include "draw/device_window.cuh"


#include "mima.cuh"
#include "../toefl/parameters.h"
#include "dg/file/json_utilities.h"

int main( int argc, char* argv[])
{
    ////Parameter initialisation ////////////////////////////////////////////
    std::stringstream title;
    Json::Value js;
    if( argc == 1)
        dg::file::file2Json( "input/default.json", js, dg::file::comments::are_discarded, dg::file::error::is_throw);
    else if( argc == 2)
        dg::file::file2Json( argv[1], js, dg::file::comments::are_discarded, dg::file::error::is_throw);
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [filename]\n";
        return -1;
    }
    std::cout<<"Point0\n";
    toefl::Parameters p( js);std::cout<<"Point1\n";
    p.display( std::cout); std::cout<<"Point2\n";
    /////////glfw initialisation ////////////////////////////////////////////
    dg::file::file2Json( "window_params.json", js, dg::file::comments::are_discarded, dg::file::error::is_throw); std::cout<<"Point2\n";
    GLFWwindow* w = draw::glfwInitAndCreateWindow( js["width"].asDouble(), js["height"].asDouble(), ""); std::cout<<"Point3\n";
    draw::RenderHostData render(js["rows"].asDouble(), js["cols"].asDouble());
    /////////////////////////////////////////////////////////////////////////
    std::cout<<"Point10\n";
    dg::x::CartesianGrid2d grid( 0, p.lx, 0, p.ly, p.n, p.Nx, p.Ny, p.bcx, p.bcy);
    //create RHS
    
    bool mhw = ( p.model == "fullF");
    mima::Mima< dg::x::CartesianGrid2d, dg::x::DMatrix, dg::x::DVec > mima( grid, p.kappa, p.tau, p.eps_pol[0], p.nu, mhw);
    dg::x::DVec one( grid.size(), 1.);
    //create initial vector
    dg::Gaussian gaussian( p.posX*grid.lx(), p.posY*grid.ly(), p.sigma, p.sigma, p.amp); //gaussian width is in absolute values
    dg::Vortex vortex( p.posX*grid.lx(), p.posY*grid.ly(), 0, p.sigma, p.amp);

//     dg::DVec phi = dg::evaluate( vortex, grid), omega( phi), y0(phi), y1(phi);
    dg::x::DVec phi = dg::evaluate( gaussian, grid), omega( phi), y0(phi), y1(phi);
    dg::Elliptic<dg::x::CartesianGrid2d, dg::x::DMatrix, dg::x::DVec> laplaceM( grid,  dg::centered);
    dg::blas2::gemv( laplaceM, phi, omega);
    dg::blas1::axpby( 1., phi, 1., omega, y0);

    dg::DVec w2d( dg::create::weights( grid));
    if( p.bcx == dg::PER && p.bcy == dg::PER)
    {
        double meanMass = dg::blas2::dot( y0, w2d, one)/(double)(p.lx*p.ly);
        std::cout << "Mean Mass is "<<meanMass<<"\n";
        dg::blas1::axpby( -meanMass, one, 1., y0);
    }
    std::string tableau;
    double rtol, atol, time = 0.;
    try{
        rtol = js["timestepper"].get("rtol", 1e-5).asDouble();
        atol = js["timestepper"].get("atol", 1e-5).asDouble();
        tableau = js[ "timestepper"].get( "tableau",
                "Bogacki-Shampine-4-2-3").asString();
    }catch ( std::exception& error){
        DG_RANK0 std::cerr << "Error in input file " << argv[1]<< std::endl;
        DG_RANK0 std::cerr << error.what() << std::endl;
        dg::abort_program();
    }
    DG_RANK0 std::cout<< "Construct timeloop ...\n";
    dg::Adaptive< dg::ERKStep< dg::DVec>> adapt(tableau, y0);

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual);
    dg::IHMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExt colors( 1.);
    //create timer
    dg::Timer t;
    double dt = 1e-5;
    unsigned itstp = js["output"]["itstp"].asUInt();
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    unsigned step = 0;
    while ( !glfwWindowShouldClose( w ))
    {
        if( p.bcx == dg::PER && p.bcy == dg::PER)
        {
            double meanMass = dg::blas2::dot( y0, w2d, one)/(double)(p.lx*p.ly);
            std::cout << "Mean Mass is "<<meanMass<<"\n";
        }
        //transform field to an equidistant grid
        dvisual = mima.potential();

        dg::assign( dvisual, hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw ions
        title << std::setprecision(2) << std::scientific;
        title <<"ne / "<<colors.scale()<<"\t";
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //transform phi
        dg::blas2::gemv( laplaceM, mima.potential(), y1);
        dg::assign( y1, hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw phi and swap buffers
        title <<"omega / "<<colors.scale()<<"\t";
        title << std::fixed;
        title << " &&   time = "<<time;
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<itstp; i++)
        {
            step++;
            if( p.bcx == dg::PER && p.bcy == dg::PER)
            {
                double meanMass = dg::blas2::dot( y0, w2d, one)/(double)(p.lx*p.ly);
                dg::blas1::axpby( -meanMass, one, 1., y0);
                meanMass = dg::blas2::dot( y0, w2d, one)/(double)(p.lx*p.ly);
                dg::blas1::axpby( -meanMass, one, 1., y0);
            }

            try{
                adapt.step( mima, time, y0, time, y0, dt, dg::pid_control,
                        dg::l2norm, rtol, atol);
            }
            catch( std::exception& fail) {
                std::cerr << "ERROR in Timestepper\n";
                std::cerr << fail.what() << std::endl;
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
        }
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
